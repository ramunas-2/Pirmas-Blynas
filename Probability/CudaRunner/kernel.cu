#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


int maxGameOverPathLengthEvolution1;

int* mask1;
int mask0;

int* matrix1;
int matrix0;

double* wonCoins;
int wonCoinsLength;

int* path1;
int path0;

int* sMask4;
int k0mask4;
int k1mask4;
int k2mask4;
int k3mask4;
int kAmask4;

int* sMask3;
int k0mask3;
int k1mask3;
int k2mask3;
int kAmask3;

int* sMask2;
int k0mask2;
int k1mask2;
int kAmask2;

int* smatrix3;
int k0matrix3;
int k1matrix3;
int k2matrix3;
int kAmatrix3;

int* smatrix2;
int k0matrix2;
int k1matrix2;
int kAmatrix2;

double* smatrixCoins2;
int k0matrixCoins2;
int k1matrixCoins2;
int kAmatrixCoins2;

int* spath2;
int k0path2;
int k1path2;
int kApath2;
















__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	//printf("Thread %d\n",i);

	long cc = 0;
	for (int i2 = 0; i2 < 1; i2++)
		for (int i1 = 0; i1 < 4000000; i1++)
		{
			cc += i1;
		}

	c[i] = cc;


}


extern "C"
{
	__declspec(dllexport) void CudaAdd(int path0, int path1[], int path2[][2][3])
	{
		//printf("GO Cuda! %d\n", length);
		//addWithCuda(c, a, b, length);

	}
}





extern "C"
{
	__declspec(dllexport) void DisplayHelloFromDLL(int length, double dArray[])
	{
		//printf("Hello from DLL !\n",x);
		for (int i = 0; i < length; i++)
		{
			dArray[i] += 4.0;
		}
	}
}

extern "C"
{
	__declspec(dllexport) void initialiseAntiPlayer2(

		int maxGameOverPathLengthEvolution1


		)
	{

	}



}



extern "C"
{
	__declspec(dllexport) void initialiseAntiPlayer(

		int _maxGameOverPathLengthEvolution1,

		int _mask1[],
		int _mask0,

		int _matrix1[],
		int _matrix0,

		double _wonCoins[],
		int _wonCoinsLength,

		int _path1[],
		int _path0,

		int _sMask4[],
		int _k0mask4,
		int _k1mask4,
		int _k2mask4,
		int _k3mask4,
		int _kAmask4,

		int _sMask3[],
		int _k0mask3,
		int _k1mask3,
		int _k2mask3,
		int _kAmask3,

		int _sMask2[],
		int _k0mask2,
		int _k1mask2,
		int _kAmask2,

		int _smatrix3[],
		int _k0matrix3,
		int _k1matrix3,
		int _k2matrix3,
		int _kAmatrix3,

		int _smatrix2[],
		int _k0matrix2,
		int _k1matrix2,
		int _kAmatrix2,

		double _smatrixCoins2[],
		int _k0matrixCoins2,
		int _k1matrixCoins2,
		int _kAmatrixCoins2,

		int _spath2[],
		int _k0path2,
		int _k1path2,
		int _kApath2

		)
	{
		printf("Hello from C++, initialising parameters.\n");

		maxGameOverPathLengthEvolution1 = _maxGameOverPathLengthEvolution1;

		mask0 = _mask0;
		mask1 = new int[mask0];
		hipMemcpy(mask1, _mask1, sizeof(int)*mask0, hipMemcpyHostToHost);

		matrix0 = _matrix0;
		matrix1 = new int[matrix0];
		hipMemcpy(matrix1, _matrix1, sizeof(int)*matrix0, hipMemcpyHostToHost);

		wonCoinsLength = _wonCoinsLength;
		wonCoins = new double[wonCoinsLength];
		hipMemcpy(wonCoins, _wonCoins, sizeof(double)*wonCoinsLength, hipMemcpyHostToHost);

		path0 = _path0;
		path1 = new int[path0];
		hipMemcpy(path1, _path1, sizeof(int)*path0, hipMemcpyHostToHost);

		k0mask4 = _k0mask4;
		k1mask4 = _k1mask4;
		k2mask4 = _k2mask4;
		k3mask4 = _k3mask4;
		kAmask4 = _kAmask4;

		sMask4 = new int[kAmask4];
		hipMemcpy(sMask4, _sMask4, sizeof(int)*kAmask4, hipMemcpyHostToHost);

		k0mask3 = _k0mask3;
		k1mask3 = _k1mask3;
		k2mask3 = _k2mask3;
		kAmask3 = _kAmask3;

		sMask3 = new int[kAmask3];
		hipMemcpy(sMask3, _sMask3, sizeof(int)*kAmask3, hipMemcpyHostToHost);

		k0mask2 = _k0mask2;
		k1mask2 = _k1mask2;
		kAmask2 = _kAmask2;

		sMask2 = new int[kAmask2];
		hipMemcpy(sMask2, _sMask2, sizeof(int)*kAmask2, hipMemcpyHostToHost);

		k0matrix3 = _k0matrix3;
		k1matrix3 = _k1matrix3;
		k2matrix3 = _k2matrix3;
		kAmatrix3 = _kAmatrix3;

		smatrix3 = new int[kAmatrix3];
		hipMemcpy(smatrix3, _smatrix3, sizeof(int)*kAmatrix3, hipMemcpyHostToHost);

		k0matrix2 = _k0matrix2;
		k1matrix2 = _k1matrix2;
		kAmatrix2 = _kAmatrix2;

		smatrix2 = new int[kAmatrix2];
		hipMemcpy(smatrix2, _smatrix2, sizeof(int)*kAmatrix2, hipMemcpyHostToHost);

		k0matrixCoins2 = _k0matrixCoins2;
		k1matrixCoins2 = _k1matrixCoins2;
		kAmatrixCoins2 = _kAmatrixCoins2;

		smatrixCoins2 = new double[kAmatrixCoins2];
		hipMemcpy(smatrixCoins2, _smatrixCoins2, sizeof(double)*kAmatrixCoins2, hipMemcpyHostToHost);

		k0path2 = _k0path2;
		k1path2 = _k1path2;
		kApath2 = _kApath2;

		spath2 = new int[kApath2];
		hipMemcpy(spath2, _spath2, sizeof(int)*kApath2, hipMemcpyHostToHost);





	}

	__declspec(dllexport) double calculateAntiPlayerExternal(double brainCells[], int allBrainCellsCount)
	{
		
		for (int i0 = 0; i0 < matrix0; i0++)
		{
			double sum = 0.0;
			for (int i1 = 0; i1 < matrix1[i0]; i1++)
			{
				double multiplication = smatrixCoins2[(i0 * k1matrix2) + i1];
				for (int i2 = 0; i2 < smatrix2[(i0 * k1matrix2) + i1]; i2++)
				{
					multiplication *= brainCells[smatrix3[(((i0 * k1matrix3) + i1) * k2matrix3) + i2]];
				}
				sum += multiplication;
			}
			wonCoins[i0] = sum;
		}
		
		int pathLength = maxGameOverPathLengthEvolution1;
		for (int i0 = 0; i0 < mask0; i0++)
		{
			for (int i1 = 0; i1 < mask1[i0]; i1++)
			{
				double bestOptionABenefit = -1000000000; //to fix
				int bestOptionAChoiceLocation = -1;
				for (int i2 = 0; i2 < sMask2[(i0 * k1mask2) + i1]; i2++)
				{
					double sumOfAllDiceCombinationsBenefit = 0;
					for (int i3 = 0; i3 < sMask3[(((i0 * k1mask3) + i1) * k2mask3) + i2]; i3++)
					{
						sumOfAllDiceCombinationsBenefit += wonCoins[sMask4[(((((i0 * k1mask4) + i1) * k2mask4) + i2) * k3mask4) + i3]];

					}
					//if (bestOptionABenefit == null || sumOfAllDiceCombinationsBenefit > bestOptionABenefit)
					if (sumOfAllDiceCombinationsBenefit > bestOptionABenefit)
					{
						bestOptionABenefit = sumOfAllDiceCombinationsBenefit;
						bestOptionAChoiceLocation = i2;
					}
				}
				for (int i2 = 0; i2 < sMask2[(i0 * k1mask2) + i1]; i2++)
				{
					for (int i3 = 0; i3 < sMask3[(((i0 * k1mask3) + i1) * k2mask3) + i2]; i3++)
					{
						double newBrainCellValue;
						if (i2 == bestOptionAChoiceLocation)
						{
							newBrainCellValue = 1.0;
						}
						else
						{
							newBrainCellValue = 0.0;
						}
						brainCells[spath2[((sMask4[(((((i0 * k1mask4) + i1) * k2mask4) + i2) * k3mask4) + i3]) * k1path2) + pathLength - 1]] = newBrainCellValue;
						wonCoins[sMask4[(((((i0 * k1mask4) + i1) * k2mask4) + i2) * k3mask4) + i3]] *= newBrainCellValue;
					}
				}
			}
			pathLength--;
		}
		double sumOfCoins = 0;
		for (int i = 0; i < wonCoinsLength; i++)
		{
			sumOfCoins += wonCoins[i];
		}
		return sumOfCoins;
	}
}



int main()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
