#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__constant__ int* maxGameOverPathLengthEvolution1;

__constant__ int* mask1;
__constant__ int* mask0;

__constant__ int* matrix1;
__constant__ int* matrix0;

__constant__ int* path1;
__constant__ int* path0;

// int* sMask4__;
// int* kAmask4__;
 //_device__ __constant__ int sMask4[10800];
__constant__ int* sMask4;

__constant__ int* k0mask4;
__constant__ int* k1mask4;
__constant__ int* k2mask4;
__constant__ int* k3mask4;
__constant__ int* kAmask4;

__constant__ int* sMask3;
__constant__ int* k0mask3;
__constant__ int* k1mask3;
__constant__ int* k2mask3;
__constant__ int* kAmask3;

__constant__ int* sMask2;
__constant__ int* k0mask2;
__constant__ int* k1mask2;
__constant__ int* kAmask2;

__constant__ int* smatrix3;
__constant__ int* k0matrix3;
__constant__ int* k1matrix3;
__constant__ int* k2matrix3;
__constant__ int* kAmatrix3;

__constant__ int* smatrix2;
__constant__ int* k0matrix2;
__constant__ int* k1matrix2;
__constant__ int* kAmatrix2;

__constant__ double* smatrixCoins2;
__constant__ int* k0matrixCoins2;
__constant__ int* k1matrixCoins2;
__constant__ int* kAmatrixCoins2;

__constant__ int* spath2;
__constant__ int* k0path2;
__constant__ int* k1path2;
__constant__ int* kApath2;
















__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	//printf("Thread %d\n",i);

	long cc = 0;
	for (int i2 = 0; i2 < 1; i2++)
		for (int i1 = 0; i1 < 4000000; i1++)
		{
			cc += i1;
		}

	c[i] = cc;


}


__global__ void antiPlayerGPU(

	double *brainCells,
	const int *allBrainCellsCount,
	double *sumOfCoins,

	const int* maxGameOverPathLengthEvolution1,

	const int* mask1,
	const int* mask0,

	const int* matrix1,
	const int* matrix0,

	const int* path1,
	const int* path0,

	const int* sMask4,
	const int* k0mask4,
	const int* k1mask4,
	const int* k2mask4,
	const int* k3mask4,
	const int* kAmask4,

	const int* sMask3,
	const int* k0mask3,
	const int* k1mask3,
	const int* k2mask3,
	const int* kAmask3,

	const int* sMask2,
	const int* k0mask2,
	const int* k1mask2,
	const int* kAmask2,

	const int* smatrix3,
	const int* k0matrix3,
	const int* k1matrix3,
	const int* k2matrix3,
	const int* kAmatrix3,

	const int* smatrix2,
	const int* k0matrix2,
	const int* k1matrix2,
	const int* kAmatrix2,

	const double* smatrixCoins2,
	const int* k0matrixCoins2,
	const int* k1matrixCoins2,
	const int* kAmatrixCoins2,

	const int* spath2,
	const int* k0path2,
	const int* k1path2,
	const int* kApath2)
{
	int iThX = threadIdx.x;
	double* wonCoins = new double[*matrix0];

	for (int i0 = 0; i0 < *matrix0; i0++)
	{
		double sum = 0.0;
		for (int i1 = 0; i1 < matrix1[i0]; i1++)
		{
			double multiplication = smatrixCoins2[(i0 * (*k1matrix2)) + i1];
			for (int i2 = 0; i2 < smatrix2[(i0 * (*k1matrix2)) + i1]; i2++)
			{
				multiplication *= brainCells[((*allBrainCellsCount)*iThX) + smatrix3[(((i0 * (*k1matrix3)) + i1) * (*k2matrix3)) + i2]];
			}
			sum += multiplication;
		}
		wonCoins[i0] = sum;
	}

	int pathLength = *maxGameOverPathLengthEvolution1;
	for (int i0 = 0; i0 < *mask0; i0++)
	{
		for (int i1 = 0; i1 < mask1[i0]; i1++)
		{
			double bestOptionABenefit = -1000000000; //to fix
			int bestOptionAChoiceLocation = -1;
			for (int i2 = 0; i2 < sMask2[(i0 * (*k1mask2)) + i1]; i2++)
			{
				double sumOfAllDiceCombinationsBenefit = 0;
				for (int i3 = 0; i3 < sMask3[(((i0 * (*k1mask3)) + i1) * (*k2mask3)) + i2]; i3++)
				{
					sumOfAllDiceCombinationsBenefit += wonCoins[sMask4[(((((i0 * (*k1mask4)) + i1) * (*k2mask4)) + i2) * (*k3mask4)) + i3]];

				}
				//if (bestOptionABenefit == null || sumOfAllDiceCombinationsBenefit > bestOptionABenefit)
				if (sumOfAllDiceCombinationsBenefit > bestOptionABenefit)
				{
					bestOptionABenefit = sumOfAllDiceCombinationsBenefit;
					bestOptionAChoiceLocation = i2;
				}
			}
			for (int i2 = 0; i2 < sMask2[(i0 * (*k1mask2)) + i1]; i2++)
			{
				for (int i3 = 0; i3 < sMask3[(((i0 * (*k1mask3)) + i1) * (*k2mask3)) + i2]; i3++)
				{
					double newBrainCellValue;
					if (i2 == bestOptionAChoiceLocation)
					{
						newBrainCellValue = 1.0;
					}
					else
					{
						newBrainCellValue = 0.0;
					}
					brainCells[((*allBrainCellsCount)*iThX) + spath2[((sMask4[(((((i0 * (*k1mask4)) + i1) * (*k2mask4)) + i2) * (*k3mask4)) + i3]) * (*k1path2)) + pathLength - 1]] = newBrainCellValue;
					wonCoins[sMask4[(((((i0 * (*k1mask4)) + i1) * (*k2mask4)) + i2) * (*k3mask4)) + i3]] *= newBrainCellValue;
				}
			}
		}
		pathLength--;
	}
	sumOfCoins[iThX] = 0;
	for (int i = 0; i < *matrix0; i++)
	{
		sumOfCoins[iThX] += wonCoins[i];
	}
	delete[] wonCoins;

}



extern "C"
{
	__declspec(dllexport) void CudaAdd(int path0, int path1[], int path2[][2][3])
	{
		//printf("GO Cuda! %d\n", length);
		//addWithCuda(c, a, b, length);

	}
}





extern "C"
{
	__declspec(dllexport) void DisplayHelloFromDLL(int length, double dArray[])
	{
		//printf("Hello from DLL !\n",x);
		for (int i = 0; i < length; i++)
		{
			dArray[i] += 4.0;
		}
	}
}

extern "C"
{
	__declspec(dllexport) void initialiseAntiPlayer2(

		int maxGameOverPathLengthEvolution1


		)
	{

	}



}



extern "C"
{
	__declspec(dllexport) void initialiseAntiPlayer(

		int _maxGameOverPathLengthEvolution1,

		int _mask1[],
		int _mask0,

		int _matrix1[],
		int _matrix0,

		int _path1[],
		int _path0,

		int _sMask4[],
		int _k0mask4,
		int _k1mask4,
		int _k2mask4,
		int _k3mask4,
		int _kAmask4,

		int _sMask3[],
		int _k0mask3,
		int _k1mask3,
		int _k2mask3,
		int _kAmask3,

		int _sMask2[],
		int _k0mask2,
		int _k1mask2,
		int _kAmask2,

		int _smatrix3[],
		int _k0matrix3,
		int _k1matrix3,
		int _k2matrix3,
		int _kAmatrix3,

		int _smatrix2[],
		int _k0matrix2,
		int _k1matrix2,
		int _kAmatrix2,

		double _smatrixCoins2[],
		int _k0matrixCoins2,
		int _k1matrixCoins2,
		int _kAmatrixCoins2,

		int _spath2[],
		int _k0path2,
		int _k1path2,
		int _kApath2

		)
	{
		printf("Hello from C++, initialising parameters.\n");

		/*
		maxGameOverPathLengthEvolution1 = new int;

		*maxGameOverPathLengthEvolution1 = 	_maxGameOverPathLengthEvolution1;

		mask0 = new int;
		*mask0=_mask0;
		mask1 = new int[*mask0];
		hipMemcpy(mask1, _mask1, sizeof(int)*(*mask0), hipMemcpyHostToHost);

		matrix0 = new int;
		*matrix0 = _matrix0;
		matrix1 = new int[*matrix0];
		hipMemcpy(matrix1, _matrix1, sizeof(int)*(*matrix0), hipMemcpyHostToHost);

		wonCoinsLength = new int;
		*wonCoinsLength = _wonCoinsLength;
		wonCoins = new double[*wonCoinsLength];
		hipMemcpy(wonCoins, _wonCoins, sizeof(double)*(*wonCoinsLength), hipMemcpyHostToHost);

		path0 = new int;
		*path0 = _path0;
		path1 = new int[*path0];
		hipMemcpy(path1, _path1, sizeof(int)*(*path0), hipMemcpyHostToHost);

		k0mask4 = new int;
		*k0mask4 = _k0mask4;
		k1mask4 = new int;
		*k1mask4 = _k1mask4;
		k2mask4 = new int;
		*k2mask4 = _k2mask4;
		k3mask4 = new int;
		*k3mask4 = _k3mask4;
		kAmask4 = new int;
		*kAmask4 = _kAmask4;

		sMask4 = new int[*kAmask4];
		hipMemcpy(sMask4, _sMask4, sizeof(int)*(*kAmask4), hipMemcpyHostToHost);

		k0mask3 = new int;
		*k0mask3 = _k0mask3;
		k1mask3 = new int;
		*k1mask3 = _k1mask3;
		k2mask3 = new int;
		*k2mask3 = _k2mask3;
		kAmask3 = new int;
		*kAmask3 = _kAmask3;

		sMask3 = new int[*kAmask3];
		hipMemcpy(sMask3, _sMask3, sizeof(int)*(*kAmask3), hipMemcpyHostToHost);

		k0mask2 = new int;
		*k0mask2 = _k0mask2;
		k1mask2 = new int;
		*k1mask2 = _k1mask2;
		kAmask2 = new int;
		*kAmask2 = _kAmask2;

		sMask2 = new int[*kAmask2];
		hipMemcpy(sMask2, _sMask2, sizeof(int)*(*kAmask2), hipMemcpyHostToHost);

		k0matrix3 = new int;
		*k0matrix3 = _k0matrix3;
		k1matrix3 = new int;
		*k1matrix3 = _k1matrix3;
		k2matrix3 = new int;
		*k2matrix3 = _k2matrix3;
		kAmatrix3 = new int;
		*kAmatrix3 = _kAmatrix3;

		smatrix3 = new int[*kAmatrix3];
		hipMemcpy(smatrix3, _smatrix3, sizeof(int)*(*kAmatrix3), hipMemcpyHostToHost);

		k0matrix2 = new int;
		*k0matrix2 = _k0matrix2;
		k1matrix2 = new int;
		*k1matrix2 = _k1matrix2;
		kAmatrix2 = new int;
		*kAmatrix2 = _kAmatrix2;

		smatrix2 = new int[*kAmatrix2];
		hipMemcpy(smatrix2, _smatrix2, sizeof(int)*(*kAmatrix2), hipMemcpyHostToHost);

		k0matrixCoins2 = new int;
		*k0matrixCoins2 = _k0matrixCoins2;
		k1matrixCoins2 = new int;
		*k1matrixCoins2 = _k1matrixCoins2;
		kAmatrixCoins2 = new int;
		*kAmatrixCoins2 = _kAmatrixCoins2;

		smatrixCoins2 = new double[*kAmatrixCoins2];
		hipMemcpy(smatrixCoins2, _smatrixCoins2, sizeof(double)*(*kAmatrixCoins2), hipMemcpyHostToHost);

		k0path2 = new int;
		*k0path2 = _k0path2;
		k1path2 = new int;
		*k1path2 = _k1path2;
		kApath2 = new int;
		*kApath2 = _kApath2;

		spath2 = new int[*kApath2];
		hipMemcpy(spath2, _spath2, sizeof(int)*(*kApath2), hipMemcpyHostToHost);
		*/



		/*
				maxGameOverPathLengthEvolution1 = new int;
				hipMemcpy(maxGameOverPathLengthEvolution1, &_maxGameOverPathLengthEvolution1, sizeof(int), hipMemcpyHostToHost);

				mask0 = new int;
				hipMemcpy(mask0, &_mask0, sizeof(int), hipMemcpyHostToHost);
				mask1 = new int[_mask0];
				hipMemcpy(mask1, _mask1, sizeof(int)*_mask0, hipMemcpyHostToHost);

				matrix0 = new int;
				hipMemcpy(matrix0, &_matrix0, sizeof(int), hipMemcpyHostToHost);
				matrix1 = new int[_matrix0];
				hipMemcpy(matrix1, _matrix1, sizeof(int)*_matrix0, hipMemcpyHostToHost);

				wonCoinsLength = new int;
				hipMemcpy(wonCoinsLength, &_wonCoinsLength, sizeof(int), hipMemcpyHostToHost);

				wonCoins = new double[_wonCoinsLength];
				hipMemcpy(wonCoins, _wonCoins, sizeof(double)*_wonCoinsLength, hipMemcpyHostToHost);

				path0 = new int;
				hipMemcpy(path0, &_path0, sizeof(int), hipMemcpyHostToHost);
				path1 = new int[_path0];
				hipMemcpy(path1, _path1, sizeof(int)*_path0, hipMemcpyHostToHost);

				k0mask4 = new int;
				hipMemcpy(k0mask4, &_k0mask4, sizeof(int), hipMemcpyHostToHost);
				k1mask4 = new int;
				hipMemcpy(k1mask4, &_k1mask4, sizeof(int), hipMemcpyHostToHost);
				k2mask4 = new int;
				hipMemcpy(k2mask4, &_k2mask4, sizeof(int), hipMemcpyHostToHost);
				k3mask4 = new int;
				hipMemcpy(k3mask4, &_k3mask4, sizeof(int), hipMemcpyHostToHost);
				kAmask4 = new int;
				hipMemcpy(kAmask4, &_kAmask4, sizeof(int), hipMemcpyHostToHost);

				sMask4 = new int[_kAmask4];
				hipMemcpy(sMask4, _sMask4, sizeof(int)*_kAmask4, hipMemcpyHostToHost);

				k0mask3 = new int;
				hipMemcpy(k0mask3, &_k0mask3, sizeof(int), hipMemcpyHostToHost);
				k1mask3 = new int;
				hipMemcpy(k1mask3, &_k1mask3, sizeof(int), hipMemcpyHostToHost);
				k2mask3 = new int;
				hipMemcpy(k2mask3, &_k2mask3, sizeof(int), hipMemcpyHostToHost);
				kAmask3 = new int;
				hipMemcpy(kAmask3, &_kAmask3, sizeof(int), hipMemcpyHostToHost);

				sMask3 = new int[_kAmask3];
				hipMemcpy(sMask3, _sMask3, sizeof(int)*_kAmask3, hipMemcpyHostToHost);

				k0mask2 = new int;
				hipMemcpy(k0mask2, &_k0mask2, sizeof(int), hipMemcpyHostToHost);
				k1mask2 = new int;
				hipMemcpy(k1mask2, &_k1mask2, sizeof(int), hipMemcpyHostToHost);
				kAmask2 = new int;
				hipMemcpy(kAmask2, &_kAmask2, sizeof(int), hipMemcpyHostToHost);

				sMask2 = new int[_kAmask2];
				hipMemcpy(sMask2, _sMask2, sizeof(int)*_kAmask2, hipMemcpyHostToHost);

				k0matrix3 = new int;
				hipMemcpy(k0matrix3, &_k0matrix3, sizeof(int), hipMemcpyHostToHost);
				k1matrix3 = new int;
				hipMemcpy(k1matrix3, &_k1matrix3, sizeof(int), hipMemcpyHostToHost);
				k2matrix3 = new int;
				hipMemcpy(k2matrix3, &_k2matrix3, sizeof(int), hipMemcpyHostToHost);
				kAmatrix3 = new int;
				hipMemcpy(kAmatrix3, &_kAmatrix3, sizeof(int), hipMemcpyHostToHost);

				smatrix3 = new int[_kAmatrix3];
				hipMemcpy(smatrix3, _smatrix3, sizeof(int)*_kAmatrix3, hipMemcpyHostToHost);

				k0matrix2 = new int;
				hipMemcpy(k0matrix2, &_k0matrix2, sizeof(int), hipMemcpyHostToHost);
				k1matrix2 = new int;
				hipMemcpy(k1matrix2, &_k1matrix2, sizeof(int), hipMemcpyHostToHost);
				kAmatrix2 = new int;
				hipMemcpy(kAmatrix2, &_kAmatrix2, sizeof(int), hipMemcpyHostToHost);

				smatrix2 = new int[_kAmatrix2];
				hipMemcpy(smatrix2, _smatrix2, sizeof(int)*_kAmatrix2, hipMemcpyHostToHost);

				k0matrixCoins2 = new int;
				hipMemcpy(k0matrixCoins2, &_k0matrixCoins2, sizeof(int), hipMemcpyHostToHost);
				k1matrixCoins2 = new int;
				hipMemcpy(k1matrixCoins2, &_k1matrixCoins2, sizeof(int), hipMemcpyHostToHost);
				kAmatrixCoins2 = new int;
				hipMemcpy(kAmatrixCoins2, &_kAmatrixCoins2, sizeof(int), hipMemcpyHostToHost);

				smatrixCoins2 = new double[_kAmatrixCoins2];
				hipMemcpy(smatrixCoins2, _smatrixCoins2, sizeof(double)*_kAmatrixCoins2, hipMemcpyHostToHost);

				k0path2 = new int;
				hipMemcpy(k0path2, &_k0path2, sizeof(int), hipMemcpyHostToHost);
				k1path2 = new int;
				hipMemcpy(k1path2, &_k1path2, sizeof(int), hipMemcpyHostToHost);
				kApath2 = new int;
				hipMemcpy(kApath2, &_kApath2, sizeof(int), hipMemcpyHostToHost);

				spath2 = new int[_kApath2];
				hipMemcpy(spath2, _spath2, sizeof(int)*_kApath2, hipMemcpyHostToHost);
				*/


		/*
		CPU_maxGameOverPathLengthEvolution1 = new int;
		hipMemcpy(maxGameOverPathLengthEvolution1, &_maxGameOverPathLengthEvolution1, sizeof(int), hipMemcpyHostToHost);

		CPU_mask0 = new int;
		hipMemcpy(mask0, &_mask0, sizeof(int), hipMemcpyHostToHost);
		CPU_mask1 = new int[_mask0];
		hipMemcpy(mask1, _mask1, sizeof(int)*_mask0, hipMemcpyHostToHost);

		CPU_matrix0 = new int;
		hipMemcpy(matrix0, &_matrix0, sizeof(int), hipMemcpyHostToHost);
		CPU_matrix1 = new int[_matrix0];
		hipMemcpy(matrix1, _matrix1, sizeof(int)*_matrix0, hipMemcpyHostToHost);

		CPU_path0 = new int;
		hipMemcpy(path0, &_path0, sizeof(int), hipMemcpyHostToHost);
		CPU_path1 = new int[_path0];
		hipMemcpy(path1, _path1, sizeof(int)*_path0, hipMemcpyHostToHost);

		CPU_k0mask4 = new int;
		hipMemcpy(k0mask4, &_k0mask4, sizeof(int), hipMemcpyHostToHost);
		CPU_k1mask4 = new int;
		hipMemcpy(k1mask4, &_k1mask4, sizeof(int), hipMemcpyHostToHost);
		CPU_k2mask4 = new int;
		hipMemcpy(k2mask4, &_k2mask4, sizeof(int), hipMemcpyHostToHost);
		CPU_k3mask4 = new int;
		hipMemcpy(k3mask4, &_k3mask4, sizeof(int), hipMemcpyHostToHost);
		CPU_kAmask4 = new int;
		hipMemcpy(kAmask4, &_kAmask4, sizeof(int), hipMemcpyHostToHost);

		CPU_sMask4 = new int[_kAmask4];
		hipMemcpy(sMask4, _sMask4, sizeof(int)*_kAmask4, hipMemcpyHostToHost);

		CPU_k0mask3 = new int;
		hipMemcpy(k0mask3, &_k0mask3, sizeof(int), hipMemcpyHostToHost);
		CPU_k1mask3 = new int;
		hipMemcpy(k1mask3, &_k1mask3, sizeof(int), hipMemcpyHostToHost);
		CPU_k2mask3 = new int;
		hipMemcpy(k2mask3, &_k2mask3, sizeof(int), hipMemcpyHostToHost);
		CPU_kAmask3 = new int;
		hipMemcpy(kAmask3, &_kAmask3, sizeof(int), hipMemcpyHostToHost);

		CPU_sMask3 = new int[_kAmask3];
		hipMemcpy(sMask3, _sMask3, sizeof(int)*_kAmask3, hipMemcpyHostToHost);

		CPU_k0mask2 = new int;
		hipMemcpy(k0mask2, &_k0mask2, sizeof(int), hipMemcpyHostToHost);
		CPU_k1mask2 = new int;
		hipMemcpy(k1mask2, &_k1mask2, sizeof(int), hipMemcpyHostToHost);
		CPU_kAmask2 = new int;
		hipMemcpy(kAmask2, &_kAmask2, sizeof(int), hipMemcpyHostToHost);

		CPU_sMask2 = new int[_kAmask2];
		hipMemcpy(sMask2, _sMask2, sizeof(int)*_kAmask2, hipMemcpyHostToHost);

		CPU_k0matrix3 = new int;
		hipMemcpy(k0matrix3, &_k0matrix3, sizeof(int), hipMemcpyHostToHost);
		CPU_k1matrix3 = new int;
		hipMemcpy(k1matrix3, &_k1matrix3, sizeof(int), hipMemcpyHostToHost);
		CPU_k2matrix3 = new int;
		hipMemcpy(k2matrix3, &_k2matrix3, sizeof(int), hipMemcpyHostToHost);
		CPU_kAmatrix3 = new int;
		hipMemcpy(kAmatrix3, &_kAmatrix3, sizeof(int), hipMemcpyHostToHost);

		CPU_smatrix3 = new int[_kAmatrix3];
		hipMemcpy(smatrix3, _smatrix3, sizeof(int)*_kAmatrix3, hipMemcpyHostToHost);

		CPU_k0matrix2 = new int;
		hipMemcpy(k0matrix2, &_k0matrix2, sizeof(int), hipMemcpyHostToHost);
		CPU_k1matrix2 = new int;
		hipMemcpy(k1matrix2, &_k1matrix2, sizeof(int), hipMemcpyHostToHost);
		CPU_kAmatrix2 = new int;
		hipMemcpy(kAmatrix2, &_kAmatrix2, sizeof(int), hipMemcpyHostToHost);

		CPU_smatrix2 = new int[_kAmatrix2];
		hipMemcpy(smatrix2, _smatrix2, sizeof(int)*_kAmatrix2, hipMemcpyHostToHost);

		CPU_k0matrixCoins2 = new int;
		hipMemcpy(k0matrixCoins2, &_k0matrixCoins2, sizeof(int), hipMemcpyHostToHost);
		CPU_k1matrixCoins2 = new int;
		hipMemcpy(k1matrixCoins2, &_k1matrixCoins2, sizeof(int), hipMemcpyHostToHost);
		CPU_kAmatrixCoins2 = new int;
		hipMemcpy(kAmatrixCoins2, &_kAmatrixCoins2, sizeof(int), hipMemcpyHostToHost);

		CPU_smatrixCoins2 = new double[_kAmatrixCoins2];
		hipMemcpy(smatrixCoins2, _smatrixCoins2, sizeof(double)*_kAmatrixCoins2, hipMemcpyHostToHost);

		CPU_k0path2 = new int;
		hipMemcpy(k0path2, &_k0path2, sizeof(int), hipMemcpyHostToHost);
		CPU_k1path2 = new int;
		hipMemcpy(k1path2, &_k1path2, sizeof(int), hipMemcpyHostToHost);
		CPU_kApath2 = new int;
		hipMemcpy(kApath2, &_kApath2, sizeof(int), hipMemcpyHostToHost);

		CPU_spath2 = new int[_kApath2];
		hipMemcpy(spath2, _spath2, sizeof(int)*_kApath2, hipMemcpyHostToHost);
		*/


		hipMalloc((void**)&maxGameOverPathLengthEvolution1, sizeof(int));
		hipMemcpy(maxGameOverPathLengthEvolution1, &_maxGameOverPathLengthEvolution1, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&mask0, sizeof(int));
		hipMemcpy(mask0, &_mask0, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&mask1, _mask0 * sizeof(int));
		hipMemcpy(mask1, _mask1, sizeof(int)*_mask0, hipMemcpyHostToDevice);

		hipMalloc((void**)&matrix0, sizeof(int));
		hipMemcpy(matrix0, &_matrix0, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&matrix1, _matrix0 * sizeof(int));
		hipMemcpy(matrix1, _matrix1, sizeof(int)*_matrix0, hipMemcpyHostToDevice);

		hipMalloc((void**)&path0, sizeof(int));
		hipMemcpy(path0, &_path0, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&path1, _path0 * sizeof(int));
		hipMemcpy(path1, _path1, sizeof(int)*_path0, hipMemcpyHostToDevice);

		hipMalloc((void**)&k0mask4, sizeof(int));
		hipMemcpy(k0mask4, &_k0mask4, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k1mask4, sizeof(int));
		hipMemcpy(k1mask4, &_k1mask4, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k2mask4, sizeof(int));
		hipMemcpy(k2mask4, &_k2mask4, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k3mask4, sizeof(int));
		hipMemcpy(k3mask4, &_k3mask4, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&kAmask4, sizeof(int));
		hipMemcpy(kAmask4, &_kAmask4, sizeof(int), hipMemcpyHostToDevice);





		hipMalloc((void**)&sMask4, _kAmask4 * sizeof(int));
		hipMemcpy(sMask4, _sMask4, sizeof(int)*_kAmask4, hipMemcpyHostToDevice);


		/*
		sMask4__ = new int[_kAmask4];
		hipMemcpy(sMask4__, _sMask4, sizeof(int)*_kAmask4, hipMemcpyHostToHost);

		
		kAmask4__ = new int;
		hipMemcpy(kAmask4__, &_kAmask4, sizeof(int), hipMemcpyHostToHost);
		*/






		


		hipMalloc((void**)&k0mask3, sizeof(int));
		hipMemcpy(k0mask3, &_k0mask3, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k1mask3, sizeof(int));
		hipMemcpy(k1mask3, &_k1mask3, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k2mask3, sizeof(int));
		hipMemcpy(k2mask3, &_k2mask3, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&kAmask3, sizeof(int));
		hipMemcpy(kAmask3, &_kAmask3, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&sMask3, _kAmask3 * sizeof(int));
		hipMemcpy(sMask3, _sMask3, sizeof(int)*_kAmask3, hipMemcpyHostToDevice);

		hipMalloc((void**)&k0mask2, sizeof(int));
		hipMemcpy(k0mask2, &_k0mask2, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k1mask2, sizeof(int));
		hipMemcpy(k1mask2, &_k1mask2, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&kAmask2, sizeof(int));
		hipMemcpy(kAmask2, &_kAmask2, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&sMask2, _kAmask2 * sizeof(int));
		hipMemcpy(sMask2, _sMask2, sizeof(int)*_kAmask2, hipMemcpyHostToDevice);

		hipMalloc((void**)&k0matrix3, sizeof(int));
		hipMemcpy(k0matrix3, &_k0matrix3, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k1matrix3, sizeof(int));
		hipMemcpy(k1matrix3, &_k1matrix3, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k2matrix3, sizeof(int));
		hipMemcpy(k2matrix3, &_k2matrix3, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&kAmatrix3, sizeof(int));
		hipMemcpy(kAmatrix3, &_kAmatrix3, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&smatrix3, _kAmatrix3 * sizeof(int));
		hipMemcpy(smatrix3, _smatrix3, sizeof(int)*_kAmatrix3, hipMemcpyHostToDevice);

		hipMalloc((void**)&k0matrix2, sizeof(int));
		hipMemcpy(k0matrix2, &_k0matrix2, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k1matrix2, sizeof(int));
		hipMemcpy(k1matrix2, &_k1matrix2, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&kAmatrix2, sizeof(int));
		hipMemcpy(kAmatrix2, &_kAmatrix2, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&smatrix2, _kAmatrix2 * sizeof(int));
		hipMemcpy(smatrix2, _smatrix2, sizeof(int)*_kAmatrix2, hipMemcpyHostToDevice);

		hipMalloc((void**)&k0matrixCoins2, sizeof(int));
		hipMemcpy(k0matrixCoins2, &_k0matrixCoins2, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k1matrixCoins2, sizeof(int));
		hipMemcpy(k1matrixCoins2, &_k1matrixCoins2, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&kAmatrixCoins2, sizeof(int));
		hipMemcpy(kAmatrixCoins2, &_kAmatrixCoins2, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&smatrixCoins2, _kAmatrixCoins2 * sizeof(double));
		hipMemcpy(smatrixCoins2, _smatrixCoins2, sizeof(double)*_kAmatrixCoins2, hipMemcpyHostToDevice);

		hipMalloc((void**)&k0path2, sizeof(int));
		hipMemcpy(k0path2, &_k0path2, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&k1path2, sizeof(int));
		hipMemcpy(k1path2, &_k1path2, sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&kApath2, sizeof(int));
		hipMemcpy(kApath2, &_kApath2, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&spath2, _kApath2 * sizeof(int));
		hipMemcpy(spath2, _spath2, sizeof(int)*_kApath2, hipMemcpyHostToDevice);



	}

	__declspec(dllexport) double calculateAntiPlayerExternal(double _brainCells[], int _allBrainCellsCount, double _sumOfCoins[], int _count)
	{


		double* brainCells;
		hipMalloc((void**)&brainCells, _allBrainCellsCount*_count*sizeof(double));
		hipMemcpy(brainCells, _brainCells, _allBrainCellsCount*_count*sizeof(double), hipMemcpyHostToDevice);

		int* allBrainCellsCount;
		hipMalloc((void**)&allBrainCellsCount, sizeof(int));
		hipMemcpy(allBrainCellsCount, &_allBrainCellsCount, sizeof(int), hipMemcpyHostToDevice);


		double* sumOfCoins;
		hipMalloc((void**)&sumOfCoins, sizeof(double)*_count);


		
		//__constant__ int* sMask4;

		//__constant__ int sMask4[10800];




		/*
		int* sMask4;

		hipMalloc((void**)&sMask4, (*kAmask4__) * sizeof(int));
		hipMemcpy(sMask4, sMask4__, sizeof(int)*(*kAmask4__), hipMemcpyHostToDevice);
		*/

		/*
		hipMemcpyToSymbol(HIP_SYMBOL(sMask4), sMask4__, sizeof(int)*(*kAmask4__), hipMemcpyHostToDevice);

		hipError_t cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(sMask4), sMask4__, sizeof(int)*(*kAmask4__), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudaMallocToSymbol failed!");
			printf("cudaMallocToSymbol failed!\n");
			printf(cudaStatus);

		}
		*/


		//antiPlayerGPU << <1, _count >> >(brainCells, allBrainCellsCount, sumOfCoins,
		antiPlayerGPU << <1, _count >> >(brainCells, allBrainCellsCount, sumOfCoins,
			maxGameOverPathLengthEvolution1,

			mask1,
			mask0,

			matrix1,
			matrix0,

			/*
			wonCoins,
			wonCoinsLength,
			*/

			path1,
			path0,

			sMask4,
			k0mask4,
			k1mask4,
			k2mask4,
			k3mask4,
			kAmask4,

			sMask3,
			k0mask3,
			k1mask3,
			k2mask3,
			kAmask3,

			sMask2,
			k0mask2,
			k1mask2,
			kAmask2,

			smatrix3,
			k0matrix3,
			k1matrix3,
			k2matrix3,
			kAmatrix3,

			smatrix2,
			k0matrix2,
			k1matrix2,
			kAmatrix2,

			smatrixCoins2,
			k0matrixCoins2,
			k1matrixCoins2,
			kAmatrixCoins2,

			spath2,
			k0path2,
			k1path2,
			kApath2

			);

		hipDeviceSynchronize();

		hipMemcpy(_sumOfCoins, sumOfCoins, _count*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(_brainCells, brainCells, _allBrainCellsCount*_count*sizeof(double), hipMemcpyDeviceToHost);

		hipFree(brainCells);
		hipFree(allBrainCellsCount);
		hipFree(sumOfCoins);

		return 0;

	}
}



int main()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
